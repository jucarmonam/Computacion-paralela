#include "hip/hip_runtime.h"
/**
 * @file main.c
 * @author Juan Pablo Carmona Muñoz (jucarmonam) - Juan Sebastian Rodríguez (juarodriguezc)
 * @date 2022-02-06
 * @copyright Copyright (c) 2022
 */

/*Para la lectura y escritura de imagenes se usan las librerias stb_image*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

#define R_ARGS 5
#define EXPORT_QUALITY 100

void initializeMatrix(int *matRGB, int width, int height, int channels, unsigned char *img)
{
    int i = 0, j = 0;
    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width; j++)
        {
            *(matRGB + (i * width + j)) = *(img + (channels * (i * width + j)));
            *(matRGB + 1 * (width * height) + (i * width + j)) = *(img + (channels * (i * width + j) + 1));
            *(matRGB + 2 * (width * height) + (i * width + j)) = *(img + (channels * (i * width + j) + 2));
        }
    }
}

void joinMatrix(int *matRGB, int width, int height, int channels, unsigned char *resImg, unsigned char *img)
{
    int i = 0, j = 0;
    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width; j++)
        {
            *(resImg + (channels * (i * width + j))) = *(matRGB + (i * width + j));
            *(resImg + (channels * (i * width + j) + 1)) = *(matRGB + 1 * (width * height) + (i * width + j));
            *(resImg + (channels * (i * width + j) + 2)) = *(matRGB + 2 * (width * height) + (i * width + j));
            if (channels == 4)
                *(resImg + (channels * (i * width + j) + 3)) = *(img + (channels * (i * width + j) + 3));
        }
    }
}

__global__ void applyFilter(int *matRGB, int *rMatRGB, int width, int height, int nThreads, int *ker)
{
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    /*Variables necesarias para la convolución*/
    int startPos = (thread_id < (width * height) % nThreads) ? ((width * height) / nThreads) * thread_id + thread_id : ((width * height) / nThreads) * thread_id + (width * height) % nThreads;
    int endPos = (thread_id < (width * height) % nThreads) ? startPos + ((width * height) / nThreads) : startPos + ((width * height) / nThreads) - 1;
    int conv = 0;

    /*Calcular la posición inicial en términos de i y j*/
    int i = (startPos / width), j = (startPos % width);

    /*Realizar la convolucion*/
    for (; startPos <= endPos; startPos++)
    {
        /*Ignorar la convolucion en los bordes*/
        if (i > 0 && i < height - 1 && j > 0 && j < width - 1)
        {
            /*Convolucion para el canal R*/
            conv = (*(ker) * *(matRGB + ((i - 1) * width + j - 1)) + *(ker + 1) * *(matRGB + ((i - 1) * width + j)) + *(ker + 2) * *(matRGB + ((i - 1) * width + j + 1)) + *(ker + 3) * *(matRGB + (i * width + j - 1)) + *(ker + 4) * *(matRGB + (i * width + j)) + *(ker + 5) * *(matRGB + (i * width + j + 1)) + *(ker + 6) * *(matRGB + ((i + 1) * width + j - 1)) + *(ker + 7) * *(matRGB + ((i + 1) * width + j)) + *(ker + 8) * *(matRGB + ((i + 1) * width + j + 1))) % 255;
            *(rMatRGB + (i * width + j)) = conv < 0 ? 0 : conv;

            /*Convolucion para el canal G*/
            conv = (*(ker) * *(matRGB + 1 * (width * height) + ((i - 1) * width + j - 1)) + *(ker + 1) * *(matRGB + 1 * (width * height) + ((i - 1) * width + j)) + *(ker + 2) * *(matRGB + 1 * (width * height) + ((i - 1) * width + j + 1)) + *(ker + 3) * *(matRGB + 1 * (width * height) + (i * width + j - 1)) + *(ker + 4) * *(matRGB + 1 * (width * height) + (i * width + j)) + *(ker + 5) * *(matRGB + 1 * (width * height) + (i * width + j + 1)) + *(ker + 6) * *(matRGB + 1 * (width * height) + ((i + 1) * width + j - 1)) + *(ker + 7) * *(matRGB + 1 * (width * height) + ((i + 1) * width + j)) + *(ker + 8) * *(matRGB + 1 * (width * height) + ((i + 1) * width + j + 1))) % 255;
            *(rMatRGB + 1 * (width * height) + (i * width + j)) = conv < 0 ? 0 : conv;

            /*Convolucion para el canal B*/
            conv = (*(ker) * *(matRGB + 2 * (width * height) + ((i - 1) * width + j - 1)) + *(ker + 1) * *(matRGB + 2 * (width * height) + ((i - 1) * width + j)) + *(ker + 2) * *(matRGB + 2 * (width * height) + ((i - 1) * width + j + 1)) + *(ker + 3) * *(matRGB + 2 * (width * height) + (i * width + j - 1)) + *(ker + 4) * *(matRGB + 2 * (width * height) + (i * width + j)) + *(ker + 5) * *(matRGB + 2 * (width * height) + (i * width + j + 1)) + *(ker + 6) * *(matRGB + 2 * (width * height) + ((i + 1) * width + j - 1)) + *(ker + 7) * *(matRGB + 2 * (width * height) + ((i + 1) * width + j)) + *(ker + 8) * *(matRGB + 2 * (width * height) + ((i + 1) * width + j + 1))) % 255;
            *(rMatRGB + 2 * (width * height) + (i * width + j)) = conv < 0 ? 0 : conv;
        }
        j += 1;
        if (j == width)
        {
            i += 1;
            j = 0;
        }
    }
}

int main(int argc, char *argv[])
{
    /*Declarar los string de lectura y escritura*/
    char *loadPath, *savePath;
    /*Variable para escoger el kernel*/
    int argKer = 0;
    /*Declaración de variable para la escritura del archivo*/
    FILE *fp;
    /*Declarar la variable para guardar la imagen*/
    unsigned char *img, *resImg;
    /*Declarar las variables necesarias para leer la imagen*/
    int width = 0, height = 0, channels = 0;
    /*Variable para el número de hilos*/
    int nThreads = 0;
    /*Variable para el número de bloques*/
    int nBlocks = 0;
    /*Crear la matriz de color, con cada uno de los canales RGB*/
    int *matRGB;
    /*Crear la matriz de color resultante, con cada uno de los canales RGB*/
    int *rMatRGB;
    /*Crear las matriz de color para el device*/
    int *d_MatRGB;
    /*Crear la matriz resultante del device*/
    int *d_rMatRGB;
    /*Crear variable para el sizeof int*/
    int size = sizeof(int);
    /*Variables necesarias para medir tiempos*/
    struct timeval tval_before, tval_after, tval_result;
    /* Error code to check return values for CUDA calls */
    hipError_t err = hipSuccess;
    /*Creación de matriz con los posibles kernel*/
    int *ker;
    /*Creación del kernel para el device*/
    int *d_ker;
    int kernels[6][9] = {
        {-1, 0, 1, -2, 0, 2, -1, 0, 1},     // Border detection (Sobel)
        {1, -2, 1, -2, 5, -2, 1, -2, 1},    // Sharpen
        {1, 1, 1, 1, -2, 1, -1, -1, -1},    // Norte
        {-1, 1, 1, -1, -2, 1, -1, 1, 1},    // Este
        {-1, -1, 0, -1, 0, 1, 0, 1, 1},     // Estampado en relieve
        {-1, -1, -1, -1, 8, -1, -1, -1, -1} // Border detection (Sobel2)
    };
    /*Verificar que la cantidad de argumentos sea la correcta*/
    if ((argc - 1) != R_ARGS)
    {
        printf("Son necesarios %d argumentos para el funcionamiento\n", R_ARGS);
        printf("Para una correcta ejecución: ./my-effect input_image output_image kernel_parameter nThreads\n");
        exit(1);
    }
    /*Cargar en las variables los parametros*/
    loadPath = *(argv + 1);
    savePath = *(argv + 2);
    argKer = atoi(*(argv + 3));
    nBlocks = atoi(*(argv + 4));
    nThreads = atoi(*(argv + 5));
    /*Verificar que el número de hilos sea válido*/
    if (nThreads <= 0 || nBlocks <= 0)
    {
        printf("El número de hilos ingresado o de bloques no es válido \n");
        exit(1);
    }
    if (argKer > 5)
    {
        printf("El parámetro de kernel debe ser menor o igual a 5 \n");
        exit(1);
    }
    /*Cargar en el ker el kernel escogido por el usuario*/
    ker = *(kernels + argKer);
    /*Cargar la imagen usando el parámetro con el nombre*/
    img = stbi_load(loadPath, &width, &height, &channels, 0);
    /*Verificar que la imagen exista y sea cargada correctamente*/
    if (img == NULL)
    {
        printf("Error al cargar la imagen \n");
        exit(1);
    }


    /*Crear cada matriz de Color dependiendo del tamaño*/

    matRGB = (int *)malloc(3 * height * width * size);
    rMatRGB = (int *)malloc(3 * height * width * size);
    if (matRGB == NULL || rMatRGB == NULL)
    {
        printf("Error al crear la matriz de colores, problema con malloc \n");
        exit(1);
    }

    /*Inicializar las matrices con los valores de la imagen*/
    initializeMatrix(matRGB, width, height, channels, img);

    /*******************************************/
    /*                  CUDA                   */
    /*******************************************/

    /*Reservar en memoria una copia del kernel de convolución*/
    err = hipMalloc((void **)&d_ker, size * 9);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_ker (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*Copiar el kernel del Host al Device*/
    err = hipMemcpy(d_ker, ker, size * 9, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy ker from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Reservar espacio en memoria para la matriz de color en device*/
    err = hipMalloc((void **)&d_MatRGB, 3 * height * width * size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device MatR (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Copiar la matriz del Host al Device*/
    err = hipMemcpy(d_MatRGB, matRGB, 3 * height * width * size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy MatR from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Crear las matrices de Color con para los resultados*/
    err = hipMalloc((void **)&d_rMatRGB, 3 * height * width * size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device MatR (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    /*Copiar la matriz del Host al Device*/
    err = hipMemcpy(d_rMatRGB, matRGB, 3 * height * width * size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy MatRGB from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Medición de tiempo de inicio*/
    gettimeofday(&tval_before, NULL);

    /*Ejecutar el kernel*/
    applyFilter<<<nBlocks, nThreads>>>(d_MatRGB, d_rMatRGB, width, height, nBlocks * nThreads, d_ker);

    /*Esperar la ejecución del kernel*/
    hipDeviceSynchronize();
    
    /*Verificar la ejecución completa*/
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch applyFilter kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Medición de tiempo de finalización*/
    gettimeofday(&tval_after, NULL);

    /*Copiar resultados del device al Host*/
    err = hipMemcpy(rMatRGB, d_rMatRGB, 3 * height * width * size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy rMatRGB from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*******************************************/

    /*Reservar el espacio de memoria para la imagen resultante*/
    resImg = (unsigned char *)malloc(width * height * channels);
    if (resImg == NULL)
    {
        printf("Error al crear la imagen, problema con malloc \n");
        exit(1);
    }

    /*Exportar la imagen resultante con el tipo de dato requerido*/
    joinMatrix(rMatRGB, width, height, channels, resImg, img);

    

    /*Guardar la imagen con el nombre especificado*/
    if (strstr(savePath, ".png"))
        stbi_write_png(savePath, width, height, channels, resImg, width * channels);
    else
        stbi_write_jpg(savePath, width, height, channels, resImg, EXPORT_QUALITY);
    /*Calcular los tiempos en tval_result*/
    timersub(&tval_after, &tval_before, &tval_result);
    /*Imprimir informe*/
    printf("------------------------------------------------------------------------------\n");
    printf("Número de bloques: %d,  Número de hilos: %d,  Imagen carga: %s\n", nBlocks, nThreads, loadPath);
    printf("Resolución: %dp,  Número de kernel (Parámetro): %d, Imagen exportada: %s\n", height, argKer, savePath);
    printf("Tiempo de ejecución: %ld.%06ld s \n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
    printf("Resumen: (RES, BLOQUES, HILOS, PARAM, TIEMPO) \t%dp\t%d\t%d\t%d\t%ld.%06ld\t\n", height, nBlocks, nThreads, argKer, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
    /* Escribir los resultados en un csv*/
    fp = fopen("times.csv", "a");
    if (fp == NULL)
    {
        printf("Error al abrir el archivo \n");
        exit(1);
    }
    fprintf(fp, "%d,%d,%d,%d,%ld.%06ld\n", height, nBlocks, nThreads, argKer, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
    fclose(fp);
    /*Liberar memoria*/
    free(matRGB);
    hipFree(d_ker);
    hipFree(d_MatRGB);
    hipFree(d_rMatRGB);
    free(resImg);
    stbi_image_free(img);
    return 0;
}