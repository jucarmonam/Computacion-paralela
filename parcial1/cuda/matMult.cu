/**
 * @file matMult.c
 * @author Juan Pablo Carmona Muñoz (jucarmonam) - Juan Sebastian Rodríguez (juarodriguezc)
 * @date 2022-0-06
 * @copyright Copyright (c) 2022
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define R_ARGS 6

/*Variable para el size del dato*/
int size = sizeof(int);

void readMatrix(char *path, char *matrixS, int n)
{
    FILE *fp;
    int i = 0;
    char c;
    fp = fopen(path, "r");
    if (fp == NULL)
    {
        printf("Error el abrir el archivo...\n");
        exit(1);
    }
    do
    {
        c = fgetc(fp);
        *(matrixS + i) = c;
        i++;
        if (feof(fp))
            break;
    } while (1);

    fclose(fp);
}

void fillMatrix(char *matS, int *matrix, int n)
{
    int i = 0;
    char *delimiter = (char *)"_";
    char *token = strtok(matS, delimiter);
    while (token != NULL)
    {
        if (i >= n * n)
        {
            printf("Las dimensiones de la matriz no coinciden con la matriz ingresada. (Más elementos) \n");
            exit(1);
        }
        *(matrix + i) = atoi(token);
        token = strtok(NULL, delimiter);
        i++;
    }
    if (i < (n * n))
    {
        printf("Las dimensiones de la matriz no coinciden con la matriz ingresada. (Menos elementos) \n");
        exit(1);
    }
}

void transpose(int *matrix, int *matrixT, int n)
{
    int i, j;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            *(matrixT + j * n + i) = *(matrix + i * n + j);
        }
    }
}

void matMult(int *A, int *B, int *C, int n)
{
    int i, j, k;
    int *BT = (int *)malloc(n * n * size);
    transpose(B, BT, n);

    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            int mult = 0;
            for (k = 0; k < n; k++)
            {
                mult += *(A + i * n + k) * *(BT + j * n + k);
            }
            *(C + i * n + j) = mult;
        }
    }
    free(BT);
}

__global__ void transposeP(int *matrix, int *matrixT, int n, int nThreads)
{
    int matrixSize = n * n;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int iter;

    int start = (thread_id < matrixSize % nThreads) ? (matrixSize / nThreads) * thread_id + thread_id : (matrixSize / nThreads) * thread_id + matrixSize % nThreads;
    int end = (thread_id < matrixSize % nThreads) ? start + (matrixSize / nThreads) : start + (matrixSize / nThreads) - 1;

    int i = (start / n), j = (start % n);

    for (iter = start; iter <= end; iter++)
    {
        *(matrixT + j * n + i) = *(matrix + i * n + j);
        j += 1;
        if (j == n)
        {
            i += 1;
            j = 0;
        }
    }
}

__global__ void matMultP(int *A, int *BT, int *C, int n, int nThreads)
{
    int matrixSize = n * n;
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x, iter;

    int start = (thread_id < matrixSize % nThreads) ? (matrixSize / nThreads) * thread_id + thread_id : (matrixSize / nThreads) * thread_id + matrixSize % nThreads;
    int end = (thread_id < matrixSize % nThreads) ? start + (matrixSize / nThreads) : start + (matrixSize / nThreads) - 1;
    int i = (start / n), j = (start % n), k;
    for (iter = start; iter <= end; iter++)
    {
        int mult = 0;
        for (k = 0; k < n; k++)
        {
            mult += *(A + i * n + k) * *(BT + j * n + k);
        }
        *(C + i * n + j) = mult;
        j += 1;
        if (j == n)
        {
            i += 1;
            j = 0;
        }
    }
}

void checkMatrix(int *matrix, int *checkMatrix, int n)
{
    int i;
    for (i = 0; i < n * n; i++)
    {
        if (*(matrix + i) != *(checkMatrix + i))
        {
            printf("Los resultados calculados en la multiplicación son diferentes \n");
            exit(1);
        }
    }
}

void writeResult(int *matrix, int n)
{
    /*Declaración de variable para la escritura del archivo*/
    FILE *fp;
    int i;
    fp = fopen("../files/matResCUDA.txt", "w+");
    if (fp == NULL)
    {
        printf("Error al leer el archivo files/matResCUDA.txt\n");
        exit(1);
    }
    for (i = 0; i < n * n; i++)
    {
        fprintf(fp, "%d", *(matrix + i));
        if (i < n * n - 1)
            fprintf(fp, "_");
    }
}

int main(int argc, char *argv[])
{
    /*Declaración de variable para la escritura del archivo*/
    FILE *fp;
    /*Variables i, j*/
    int i, j;
    /*Arreglo para la mariz A, B y C*/
    int *A, *B, *C, *BT, *ChkRes;
    /*Crear las matrices en el device*/
    int *d_A, *d_B, *d_C, *d_BT;
    /*Variables para el PATH de matA y matB*/
    char *pathA, *pathB;
    /*Variable para la matriz A como String*/
    char *matrixAS;
    /*Variable para la matriz B como String*/
    char *matrixBS;
    /*Variable para la dimensión N de la matriz y el tamaño máximo de la matriz*/
    int n, maxSize;
    /*Variable para el número de hilos*/
    int nThreads = 0;
    /*Variable para el número de bloques*/
    int nBlocks = 0;
    /*Variable para el número de cifras del N */
    int numSize;
    /*Variable de automatización*/
    int test;
    /* Error code to check return values for CUDA calls */
    hipError_t err = hipSuccess;
    /*Variables necesarias para medir tiempos*/
    struct timeval tval_before, tval_after, tval_result;
    /*Verificar que el número de argumentos sea correcto*/
    if ((argc - 1) != R_ARGS)
    {
        printf("Son necesarios %d argumentos para el funcionamiento\n", R_ARGS);
        printf("Para una correcta ejecución: ./matMult pathMatrixA pathMatrixB n nBlocks nThreads testing\n");
        exit(1);
    }

    /*Cargar en las variables los parametros*/
    pathA = *(argv + 1);
    pathB = *(argv + 2);
    n = atoi(*(argv + 3));
    numSize = (int)strlen(*(argv + 3));
    nBlocks = atoi(*(argv + 4));
    nThreads = atoi(*(argv + 5));
    test = atoi(*(argv + 6));
    /*Verificar que el número de hilos y bloques sea válido*/
    if (nThreads <= 0 || nBlocks <= 0)
    {
        printf("El número de hilos ingresado o de bloques no es válido \n");
        exit(1);
    }

    if (n < 2)
    {
        printf("La matriz debe ser de tamaño mayor o igual a 2 * 2 \n");
        exit(1);
    }

    /*Definir el tamaño maximo dependiendo del N de la matriz*/
    maxSize = n * n * (numSize + 1 + 1);

    /*Reservar el espacio para las matrices*/
    matrixAS = (char *)malloc(maxSize);
    matrixBS = (char *)malloc(maxSize);

    /*Cargar las matrices del PATH*/
    readMatrix(pathA, matrixAS, n);
    readMatrix(pathB, matrixBS, n);

    /*Reservar en memoria el espacio para las matrices*/
    A = (int *)malloc(n * n * size);
    B = (int *)malloc(n * n * size);
    BT = (int *)malloc(n * n * size);
    C = (int *)malloc(n * n * size);
    ChkRes = (int *)malloc(n * n * size);

    if (A == NULL || B == NULL || C == NULL || BT == NULL || ChkRes == NULL)
    {
        printf("Error al crear las matrices, error en la ejecución de malloc \n");
        exit(1);
    }

    /*Se llenan las matrices con los valores almacenados en los String*/
    fillMatrix(matrixAS, A, n);
    fillMatrix(matrixBS, B, n);

    /*Realizar la multiplicación con el algoritmo secuencial para verificar*/
    matMult(A, B, ChkRes, n);

    /* Escribir los resultados en un csv*/
    fp = fopen("../files/timesCUDA.csv", "a");
    if (fp == NULL)
    {
        printf("Error al abrir el archivo csv \n");
        exit(1);
    }

    printf("------------------------------------------------\n");
    printf("                      CUDA                      \n");
    printf("------------------------------------------------\n");
    printf("              Matrix  %d x %d                   \n", n, n);
    printf("------------------------------------------------\n");

    /*Reservar en memoria una copia de la matriz A*/
    err = hipMalloc((void **)&d_A, size * n * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Copiar la matriz A del Host al Device*/
    err = hipMemcpy(d_A, A, size * n * n, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Reservar en memoria una copia de la matrix B*/
    err = hipMalloc((void **)&d_B, size * n * n);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Copiar matrix B del Host al Device*/
    err = hipMemcpy(d_B, B, size * n * n, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Reservar espacio en memoria para la matriz transpuesta de B en device*/
    err = hipMalloc((void **)&d_BT, n * n * size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_BT (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*Reservar espacio en memoria para la matriz de resultados*/
    err = hipMalloc((void **)&d_C, n * n * size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*En caso de que el parametro de test sea 0, se realiza una única prueba*/
    if (test == 0)
    {
        /*Medición de tiempo de inicio*/
        gettimeofday(&tval_before, NULL);

        /*Realizar la transposición*/
        transposeP<<<nBlocks, nThreads>>>(d_B, d_BT, n, nBlocks * nThreads);

        hipDeviceSynchronize();

        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch transposeP (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        /*Realizar la multiplicación de matrices*/
        matMultP<<<nBlocks, nThreads>>>(d_A, d_BT, d_C, n, nBlocks * nThreads);

        hipDeviceSynchronize();

        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch matMultP (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        /*Medición de tiempo de finalización*/
        gettimeofday(&tval_after, NULL);

        /* Copiar el resultado de vuelta al host*/
        err = hipMemcpy(C, d_C, size * n * n, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        checkMatrix(C, ChkRes, n);

        /*Calcular los tiempos en tval_result*/
        timersub(&tval_after, &tval_before, &tval_result);

        printf("Tiempo de ejecución ( %d bloques, %d hilos ): %ld.%06ld s \n", nBlocks, nThreads, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
        fprintf(fp, "%d,%d,%d,%ld.%06ld\n", n, nBlocks, nThreads, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
    }
    else
    {
        for (i = 1; i <= nBlocks; i *= 2)
        {
            printf("------------------------------------------------\n");
            for (j = 1; j <= nThreads; j += 20)
            {
                /*Medición de tiempo de inicio*/
                gettimeofday(&tval_before, NULL);

                /*Paralelizar el algoritmo*/
                transposeP<<<i, j>>>(d_B, d_BT, n, i * j);

                hipDeviceSynchronize();

                err = hipGetLastError();

                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to launch transposeP (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }

                matMultP<<<i, j>>>(d_A, d_BT, d_C, n, i * j);

                hipDeviceSynchronize();

                err = hipGetLastError();

                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to launch matMultP (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }

                /*Medición de tiempo de finalización*/
                gettimeofday(&tval_after, NULL);

                /*Calcular los tiempos en tval_result*/
                timersub(&tval_after, &tval_before, &tval_result);

                printf("Tiempo de ejecución ( %d bloques , %d hilos ): %ld.%06ld s \n", i, j, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
                fprintf(fp, "%d,%d,%d,%ld.%06ld\n", n, i, j, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

                // Copiar el resultado de vuelta en el host
                err = hipMemcpy(C, d_C, size * n * n, hipMemcpyDeviceToHost);
                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to copy C from device to host (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }
                checkMatrix(C, ChkRes, n);
                /*Probar caso base*/
                if (j == 1)
                    j = -10;
            }
        }
        i /= 2;
        if (nBlocks > i)
        {
            i = nBlocks;
            printf("------------------------------------------------\n");
            for (j = 10; j <= nThreads; j += 20)
            {
                /*Medición de tiempo de inicio*/
                gettimeofday(&tval_before, NULL);

                /*Paralelizar el algoritmo*/
                transposeP<<<i, j>>>(d_B, d_BT, n, i * j);

                hipDeviceSynchronize();

                err = hipGetLastError();

                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to launch transposeP (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }

                matMultP<<<i, j>>>(d_A, d_BT, d_C, n, i * j);

                hipDeviceSynchronize();

                err = hipGetLastError();

                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to launch matMultP (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }

                /*Medición de tiempo de finalización*/
                gettimeofday(&tval_after, NULL);

                /*Calcular los tiempos en tval_result*/
                timersub(&tval_after, &tval_before, &tval_result);

                printf("Tiempo de ejecución ( %d bloques , %d hilos ): %ld.%06ld s \n", i, j, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
                fprintf(fp, "%d,%d,%d,%ld.%06ld\n", n, i, j, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

                // Copiar el resultado de vuelta en el host
                err = hipMemcpy(C, d_C, size * n * n, hipMemcpyDeviceToHost);
                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to copy C from device to host (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }
                checkMatrix(C, ChkRes, n);
            }
        }
    }
    fclose(fp);
    writeResult(C, n);
    /*Liberar memoria*/
    free(A);
    free(B);
    free(C);
    free(BT);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_BT);
    return 0;
}